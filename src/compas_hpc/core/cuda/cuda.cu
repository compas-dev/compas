

#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void add(float *a, float *b, float *c)
{
    int id = threadIdx.x;

    c[id] = a[id] + b[id];
}


int main()
{
    float a[] = {1., 2., 3.};
    float b[] = {4., 5., 6.};
    float c[3];
    float *a_;
    float *b_;
    float *c_;

    int size = 3 * sizeof(float);

    hipMalloc((void**) &a_, size);
    hipMalloc((void**) &b_, size);
    hipMalloc((void**) &c_, size);

    hipMemcpy(a_, a, size, hipMemcpyHostToDevice);
    hipMemcpy(b_, b, size, hipMemcpyHostToDevice);

    dim3 dimGrid(1, 1, 1);
    dim3 dimBlock(3, 1, 1);
    add <<< dimGrid, dimBlock >>> (a_, b_, c_);

    hipMemcpy(c, c_, size, hipMemcpyDeviceToHost);

    hipFree(a_);
    hipFree(b_);
    hipFree(c_);

    printf("%f %f %f\n", c[0], c[1], c[2]);
}
